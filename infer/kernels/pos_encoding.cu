// adapted from https://github.com/vllm-project/vllm/blob/b9fe4616f98b77b4b9458bce203aa6544cb31ef2/csrc/pos_encoding_kernels.cu


#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <assert.h>
#include <stdio.h>


template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
  scalar_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = __ldg(cos_ptr + x_index);
    sin = __ldg(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = __ldg(cos_ptr + x_index / 2);
    sin = __ldg(sin_ptr + x_index / 2);
  }

  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template<typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
  const int64_t* __restrict__ positions,        // [num_tokens]
  scalar_t* __restrict__ query,                 // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int query_stride,
  const int key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    if (token_idx == 0 &&
    token_head + rot_offset + embed_dim >= query_stride
    ) {
      // token_head: 3968, rot_offset: 63, embed_dim: 128, query_stride: 4096

      printf("i: %d, head_idx: %d, head_size: %d, token_head: %d, rot_offset: %d, embed_dim: %d, query_stride: %d\n",
      i, head_idx, head_size,
      token_head, rot_offset, embed_dim, query_stride);
      assert(false);
    }
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

#define scalar_t __hip_bfloat16

extern "C" void rotary_embedding_bf16(
  const int64_t* __restrict__ positions,        // [num_tokens]
  scalar_t* __restrict__ query,                 // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int32_t num_tokens,
  const int32_t rot_dim,
  const int32_t query_stride,
  const int32_t key_stride,
  const int32_t num_heads,
  const int32_t num_kv_heads,
  const int32_t head_size)
{

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const hipStream_t stream = 0; // Use the default stream.
  // const cudaStream_t stream = at::cuda::getCurrentCUDAStream();
  rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
    positions, query, key, cos_sin_cache, rot_dim, query_stride, key_stride, num_heads, num_kv_heads, head_size);
}
